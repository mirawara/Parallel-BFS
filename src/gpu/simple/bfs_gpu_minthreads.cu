#include "hip/hip_runtime.h"
#include "bfs_gpu.cuh"

using namespace std;

#define DEBUG(x)
#define N_THREADS_PER_BLOCK 32


__global__
void computeNextQueueOpt(int *adjacencyList, int *edgesOffset, int *edgesSize, int *distance,
                         int queueSize, int *currentQueue, int *nextQueueSize, int *nextQueue, int level) {

    /* Id used to locate a thread in the grid */
    const int node_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    /* Check */
    if (node_id >= queueSize)
        return;

    /* Each column works on a specific node */
    int current = currentQueue[node_id];

    /* Number of neighbors */
    int i = edgesOffset[current];

    if (col < edgesSize[current]) {

        /* Every thread on a column visit a neighbor */
        int v = adjacencyList[i + col];

        if (distance[v] == INT_MAX) {

            distance[v] = level + 1;
            int position = atomicAdd(nextQueueSize, 1);
            nextQueue[position] = v;

        }
        
    }
    

}


void bfsGPUMethod1Opt(int start, Graph &G, vector<int> &distance, vector<bool> &visited, std::ofstream *file, int block_size) {

    /* Max number of neighbors */
    int max = *max_element(G.edgesSize.begin(), G.edgesSize.end());

    /* Initialization of GPU variables */
    int *d_adjacencyList;
    int *d_edgesOffset;
    int *d_edgesSize;
    int *d_firstQueue;
    int *d_secondQueue;
    int *d_nextQueueSize;
    int *d_distance; // output


    /* Initialization of CPU variables */
    int currentQueueSize = 1;
    const int NEXT_QUEUE_SIZE = 0;
    int level = 0;

    /* Allocation on device */
    const int size = G.numVertices * sizeof(int);
    const int adjacencySize = G.adjacencyList.size() * sizeof(int);
    hipMalloc((void **) &d_adjacencyList, adjacencySize);
    hipMalloc((void **) &d_edgesOffset, size);
    hipMalloc((void **) &d_edgesSize, size);
    hipMalloc((void **) &d_firstQueue, size);
    hipMalloc((void **) &d_secondQueue, size);
    hipMalloc((void **) &d_distance, size);
    hipMalloc((void **) &d_nextQueueSize, sizeof(int));


    /* Copy to device */
    hipMemcpy(d_adjacencyList, &G.adjacencyList[0], adjacencySize, hipMemcpyHostToDevice);
    hipMemcpy(d_edgesOffset, &G.edgesOffset[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_edgesSize, &G.edgesSize[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_firstQueue, &start, sizeof(int), hipMemcpyHostToDevice);

    /* Record performance */
    hipEvent_t startEv, stopEv;
    hipEventCreate(&startEv);
    hipEventCreate(&stopEv);
    hipEventRecord(startEv);

    /* Initialize and copy output */
    distance = vector<int>(G.numVertices, INT_MAX);
    distance[start] = 0;
    hipMemcpy(d_distance, distance.data(), size, hipMemcpyHostToDevice);

    //int block_size = 10;

    /* Compute the optimized resources
     * --> Explained below */
    int y_grid = 1;
    int y_dim;

    if (max <= block_size)
        y_dim = max;
    else {
        y_dim = block_size;
        y_grid = ceil((double) max / (double) block_size);
    }

    int* d_currentQueue;
    int* d_nextQueue;

    d_currentQueue = d_firstQueue;
    d_nextQueue = d_secondQueue;

    while (currentQueueSize > 0) {

        /* Allocate the exact number of resources used =>
         * A column for each node to visit and
         * a row for the maximum number of neighbors */
        int x_grid = 1;
        int x_dim;
        if (currentQueueSize <= block_size)
            x_dim = currentQueueSize;
        else {
            x_dim = block_size;
            x_grid = ceil((double) currentQueueSize / (double) block_size);
        }
        dim3 grid(x_grid, y_grid);
        dim3 block(x_dim, y_dim);

        /* Call the kernel with the optimized resources */
        computeNextQueueOpt<<<grid, block>>>(d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance,
                                             currentQueueSize, d_currentQueue, d_nextQueueSize,
                                             d_nextQueue, level);

        /* Increment of the level*/
        level++;

        if (level % 2 == 0) {
            d_currentQueue = d_firstQueue;
            d_nextQueue = d_secondQueue;
        }
        else {
            d_currentQueue = d_secondQueue;
            d_nextQueue = d_firstQueue;
        }

        hipDeviceSynchronize();

        /* New queues size */
        hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
    }

    /* Copy the output */
    hipMemcpy(&distance[0], d_distance, size, hipMemcpyDeviceToHost);

    /* Record performance */
    hipEventRecord(stopEv);
    hipDeviceSynchronize();
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, startEv, stopEv);

    /* Cleanup */
    hipFree(d_adjacencyList);
    hipFree(d_edgesOffset);
    hipFree(d_edgesSize);
    hipFree(d_firstQueue);
    hipFree(d_secondQueue);
    hipFree(d_distance);

    /* Write performance on file */
    *file << elapsed << ";" << endl;
}
