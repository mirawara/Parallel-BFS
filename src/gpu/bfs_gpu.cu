#include "hip/hip_runtime.h"
#include "bfs_gpu.cuh"

using namespace std;

#define DEBUG(x)
#define N_THREADS_PER_BLOCK 32

__global__
void computeNextQueue(int *adjacencyList, int *edgesOffset, int *edgesSize, int *distance,
                      int queueSize, int *currentQueue, int *nextQueueSize, int *nextQueue, int level) {
    /* Thread ID */
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    /* Check */
    if (tid < queueSize) {

        /* Every node in the queue is assigned to a specidic thread */
        int current = currentQueue[tid];

        /* Check for unvisited neighbors */
        for (int i = edgesOffset[current]; i < edgesOffset[current] + edgesSize[current]; ++i) {

            int v = adjacencyList[i];

            if (distance[v] == INT_MAX) { /* => unvisited */

                /* Mark with the distance from the starting point */
                distance[v] = level + 1;

                /* Increment the shared variable (necessary) */
                int position = atomicAdd(nextQueueSize, 1);

                /* Add the node in the next queue */
                nextQueue[position] = v;
            }
           // __syncwarp();
        }
    }
    
}


void bfsGPU(int start, Graph &G, vector<int> &distance, vector<bool> &visited, std::ofstream *file, int block_size) {

    const int n_blocks = (G.numVertices + block_size - 1) / block_size;

    /* Initialization of GPU variables */
    int *d_adjacencyList;
    int *d_edgesOffset;
    int *d_edgesSize;
    int *d_firstQueue;
    int *d_secondQueue;
    int *d_nextQueueSize;
    int *d_distance; // output


    /* Initialization of CPU variables */
    int currentQueueSize = 1;
    const int NEXT_QUEUE_SIZE = 0;
    int level = 0;

    /* Allocation on device */
    const int size = G.numVertices * sizeof(int);
    const int adjacencySize = G.adjacencyList.size() * sizeof(int);
    hipMalloc((void **) &d_adjacencyList, adjacencySize);
    hipMalloc((void **) &d_edgesOffset, size);
    hipMalloc((void **) &d_edgesSize, size);
    hipMalloc((void **) &d_firstQueue, size);
    hipMalloc((void **) &d_secondQueue, size);
    hipMalloc((void **) &d_distance, size);
    hipMalloc((void **) &d_nextQueueSize, sizeof(int));

    /* Copy to device */
    hipMemcpy(d_adjacencyList, &G.adjacencyList[0], adjacencySize, hipMemcpyHostToDevice);
    hipMemcpy(d_edgesOffset, &G.edgesOffset[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_edgesSize, &G.edgesSize[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_firstQueue, &start, sizeof(int), hipMemcpyHostToDevice);

    /* Start measuring execution time */
    hipEvent_t startEv, stopEv;
    hipEventCreate(&startEv);
    hipEventCreate(&stopEv);
    hipEventRecord(startEv);

    /* Initiale and copy the output */
    distance = vector<int>(G.numVertices, INT_MAX);
    distance[start] = 0;
    hipMemcpy(d_distance, distance.data(), size, hipMemcpyHostToDevice);

    /*
    d_currentQueue = d_firstQueue;
    d_nextQueue = d_secondQueue;*/

    /* While there are node to visit */
    while (currentQueueSize > 0) {

        int* d_currentQueue;
        int* d_nextQueue;

        /* The next queue of the previous step becomes the current queue */
        if (level % 2 == 0) {
            d_currentQueue = d_firstQueue;
            d_nextQueue = d_secondQueue;
        }
        else {
            d_currentQueue = d_secondQueue;
            d_nextQueue = d_firstQueue;
        }

        /* Calling the kernel with block of 32x32 threads, at least #vertices threads in total */
        computeNextQueue<<<n_blocks, block_size>>>(d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance,
                                                            currentQueueSize, d_currentQueue, d_nextQueueSize,
                                                            d_nextQueue, level);

        hipDeviceSynchronize();

        /* Increment of the level */
        ++level;

        /* New queues size */
        hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
    }

    /* Retrieve the output */
    hipMemcpy(&distance[0], d_distance, size, hipMemcpyDeviceToHost);

    /* Measuring performance */
    hipEventRecord(stopEv);
    hipDeviceSynchronize();
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, startEv, stopEv);

    /* Cleanup */
    hipFree(d_adjacencyList);
    hipFree(d_edgesOffset);
    hipFree(d_edgesSize);
    hipFree(d_firstQueue);
    hipFree(d_secondQueue);
    hipFree(d_distance);

    /* Write performance on file */
    *file << elapsed << ";";
}
